#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/group_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void GroupNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
  if (bottom[0] != top[0]) {
    caffe_copy(bottom[0]->count(), bottom_data, top_data);
  }

  // compute mean
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * group_num_, chip_num_ * spatial_dim_,
        1. / (chip_num_ * spatial_dim_), bottom_data,cube_sum_multiplier_.gpu_data(), 0.,
        mean_.mutable_gpu_data());

  // subtract mean
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,num_ * group_num_,
      chip_num_ * spatial_dim_, 1, -1, mean_.gpu_data(),
      cube_sum_multiplier_.gpu_data(), 1., top_data);
 
  // compute variance using var(X) = E((X-EX)^2)
  caffe_gpu_mul(top[0]->count(), top[0]->gpu_data(), top[0]->gpu_data(),temp_.mutable_gpu_data());  // (X-EX)^2
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * group_num_, chip_num_ * spatial_dim_,
        1. / (chip_num_ * spatial_dim_), temp_.gpu_data(),
        cube_sum_multiplier_.gpu_data(), 0.,variance_.mutable_gpu_data());
  
  // normalize variance
  caffe_gpu_add_scalar(variance_.count(), eps_, variance_.mutable_gpu_data());
  caffe_gpu_sqrt(variance_.count(), variance_.gpu_data(),variance_.mutable_gpu_data());

  // div variance    
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * group_num_,
      chip_num_ * spatial_dim_, 1, 1., variance_.gpu_data(),
      cube_sum_multiplier_.gpu_data(), 0., temp_.mutable_gpu_data());

  caffe_gpu_div(top[0]->count(), top_data, temp_.gpu_data(), top_data);
  
  // TODO(cdoersch): The caching is only needed because later in-place layers
  //                 might clobber the data.  Can we skip this if they won't?
  
  caffe_copy(x_norm_.count(), top_data, x_norm_.mutable_gpu_data());
}

template <typename Dtype>
void GroupNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  
  const Dtype* top_data = x_norm_.gpu_data();
  const Dtype* top_diff;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  if (bottom[0] != top[0]) {
    top_diff = top[0]->gpu_diff();
  } else {
    caffe_copy(x_norm_.count(), top[0]->gpu_diff(), x_norm_.mutable_gpu_diff());
    top_diff = x_norm_.gpu_diff();
  }
 
  // if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
  //
  // dE(Y)/dX =
  //   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
  //     ./ sqrt(var(X) + eps)
  //
  // where \cdot and ./ are hadamard product and elementwise division,
  // respectively, dE/dY is the top diff, and mean/var/sum are all computed
  // along all dimensions except the channels dimension.  In the above
  // equation, the operations allow for expansion (i.e. broadcast) along all
  // dimensions except the channels dimension where required.
 
  // sum(dE/dY \cdot Y)
  caffe_gpu_mul(temp_.count(), top_data, top_diff, bottom_diff);
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * group_num_, chip_num_ * spatial_dim_, 1.,
      bottom_diff, cube_sum_multiplier_.gpu_data(), 0.,
      mean_.mutable_gpu_data());
  
  // reshape (broadcast) the above
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * group_num_,
      chip_num_ * spatial_dim_, 1, 1., mean_.gpu_data(),
      cube_sum_multiplier_.gpu_data(), 0., bottom_diff);

  // sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_mul(temp_.count(), top_data, bottom_diff, bottom_diff);

  // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * group_num_, chip_num_ * spatial_dim_, 1.,
      top_diff, cube_sum_multiplier_.gpu_data(), 0.,
      mean_.mutable_gpu_data());
 
  // reshape (broadcast) the above to make
  // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * group_num_,
      chip_num_ * spatial_dim_, 1, 1., mean_.gpu_data(),
      cube_sum_multiplier_.gpu_data(), 1., bottom_diff);

  // dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y
  caffe_gpu_axpby(temp_.count(), Dtype(1), top_diff,
      Dtype(-1. / (chip_num_ * spatial_dim_)), bottom_diff);

  // note: temp_ still contains sqrt(var(X)+eps), computed during the forward
  // pass.
  caffe_gpu_div(temp_.count(), bottom_diff, temp_.gpu_data(), bottom_diff);
}

 

INSTANTIATE_LAYER_GPU_FUNCS(GroupNormLayer);


}  // namespace caffe




